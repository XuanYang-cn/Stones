/*
   Copyright 2019 Cleuton Sampaio
   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at
     http://www.apache.org/licenses/LICENSE-2.0
   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vecmul(float *A, float* B, float *C, int size)
{
    // Row and Column indexes:
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    // Are they bellow the maximum?
    if (col < size && row < size) {
       float result = 0;
       for(int ix=0;ix<size;ix++) {
          result += A[row*size+ix]*B[ix*size+col];
       }
       C[row*size+col] = result;
    }
}

extern "C" {

    void maxmul(float *A, float* B, float *C, int size) {

        int total = size*size;

        // Allocate device memory:
        float* gpu_A;
        float* gpu_B;
        float* gpu_C;
        int msize = total * sizeof(float);
        hipMalloc((void**)&gpu_A, msize);
        hipMemcpy(gpu_A,A,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_B, msize);
        hipMemcpy(gpu_B,B,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_C,msize);

        // Blocks & grids:
        dim3 blocks(size,size);
        dim3 grid(1,1);

        // Call the kernel:
        vecmul<<<grid,blocks>>>(gpu_A,gpu_B,gpu_C,size);

        // Get the result Matrix:
        hipMemcpy(C,gpu_C,msize,hipMemcpyDeviceToHost);

        //Free device matrices
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
    }

}
